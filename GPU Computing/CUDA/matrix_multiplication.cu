#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
int N = 0;

__global__
void dot( int *a, int ra, int ca, int *b, int rb, int cb, int *c )
{
   // shared memory for reesults of multiplication
   __shared__ int temp[N];

   // __shared__ int globalSum;
   
   for( int i = 0;i<ra;i++)
   {
		for(int j=0;j<cb;j++)
		{
			temp[threadIdx.x] = a[i*ca + threadIdx.x] * b[threadIdx.x*cb + j];
			__syncthreads( );


		   // Thread 0 sums the pairwise products
		   if ( threadIDx.x == 0 )
		   {
			  int sum = 0;
		 
			  for( int i = 0; i < N; ++i )
				 sum += temp[i];

			  c[i*cb + j] = sum;

		   }
		}
   }

   
   
}
 

void cpu_matrix_multiplication(int *A, int *B, int *C, int ra, int ca, int rb, int cb)
{
	for(int i=0 ; i < ra;i++)
	{
		for( int j=0; j < cb; j++)
		{
			C[i*cb + j] = 0;
			for(int k=0; k<ca;k++)
			{
				C[i*cb + j] += A[i*ca + k] + B[ k * cb + j];
			}
		}
	}
} 
 
 
 
void fillMatrix(int *mat, int rows, int columns)
{
	for( int i = 0; i < rows; i++ )
    {
		for(int j = 0; j < columns ; j++)
		{
			mat[i*columns + j] = rand( ) % 10;
		}
    }
}
 
void printMatrix(int *mat,int rows, int columns)
{
	for( int i = 0; i < rows; i++ )
    {
		for(int j = 0; j < columns ; j++)
		{
			printf("%d ",mat[i*columns + j]);
		}
		printf("\n");
    }
}
 
int main( void )
{

   int *a, *b, *c;			// host copies of a, b, c
   int *dev_a, *dev_b, *dev_c;		// device copies of a, b, c
   
   /*  My Code */
	//int size = N * sizeof( int );	// space for N integers

	
	int ra = 3;
	int ca = 3;
	
	N = ca;
	
	int a_size = ra * ca * sizeof(int);
	
	int rb = 3;
	int cb = 3;
	
	int b_size = rb * cb * sizeof(int);
	
	int c_size = ra * cb * sizeof(int); 
   
   
   

   // allocate host copies of a, b, c
   a = ( int * ) malloc( a_size );
   b = ( int * ) malloc( b_size );
   c = ( int * ) calloc( c_size );
   d = ( int * ) calloc( c_size );

   // allocate device copies of a, b, c
   hipMalloc( ( void** ) &dev_a, a_size );
   hipMalloc( ( void** ) &dev_b, b_size );
   hipMalloc( ( void** ) &dev_c, c_size);

   // initialize host copies of a, b
   fillMatrix(a,ra,ca);
   fillMatrix(b,rb,cb);

   //printing Matrices
   printf("Matrix A: \n");
   printMatrix(a,ra,ca);
   printf("\n");
   
   printf("Matrix B: \n");
   printMatrix(b,rb,cb);
   printf("\n");
   
   // copy inputs to device
   hipMemcpy( dev_a, a, a_size, hipMemcpyHostToDevice );
   hipMemcpy( dev_b, b, b_size, hipMemcpyHostToDevice );

   // launch add( ) kernel on GPU, passing parameters
   dot<<< 1, N >>>( dev_a, ra, ca, dev_b, rb, cb, dev_c );

   // copy device result back to host copy of c
   hipMemcpy( c, dev_c, sizeof( int ), hipMemcpyDeviceToHost );
   
   printf("Matrix C: \n");
   printMatrix(c,ra,cb);
   printf("\n");
   
   cpu_matrix_multiplication(a,b,d,ra,ca,rb,cb);

   printf("Matrix D: \n");
   printMatrix(d,ra,cb);
   printf("\n");
   
   
   // deallocate host copies of a, b, c
   free( a );
   free( b );
   free( c );

   // deallocate device copies of a, b, c
   hipFree( dev_a ); 
   hipFree( dev_b );
   hipFree( dev_c );

   return 0;

}
